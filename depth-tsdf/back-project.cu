#include "hip/hip_runtime.h"
// ---------------------------------------------------------
// Author: Andy Zeng, Princeton University, 2016
// ---------------------------------------------------------

#include <iostream>
#include <fstream>
#include <iomanip>
#include <sstream>
#include <string>
#include "utils.hpp"

// CUDA kernel function to integrate a TSDF voxel volume given depth images
__global__
void Integrate(float * cam_K, float * cam2base, float * depth_im,
               int im_height, int im_width, int voxel_grid_dim_x, int voxel_grid_dim_y, int voxel_grid_dim_z,
               float voxel_grid_origin_x, float voxel_grid_origin_y, float voxel_grid_origin_z, float voxel_size, float trunc_margin,
               float * voxel_grid_TSDF) {

  int pt_grid_z = blockIdx.x;
  int pt_grid_y = threadIdx.x;

  for (int pt_grid_x = 0; pt_grid_x < voxel_grid_dim_x; ++pt_grid_x) {

    // Convert voxel center from grid coordinates to base frame camera coordinates
    float pt_base_x = voxel_grid_origin_x + pt_grid_x * voxel_size;
    float pt_base_y = voxel_grid_origin_y + pt_grid_y * voxel_size;
    float pt_base_z = voxel_grid_origin_z + pt_grid_z * voxel_size;

    // Convert from base frame camera coordinates to current frame camera coordinates
    float tmp_pt[3] = {0};
    tmp_pt[0] = pt_base_x - cam2base[0 * 4 + 3];
    tmp_pt[1] = pt_base_y - cam2base[1 * 4 + 3];
    tmp_pt[2] = pt_base_z - cam2base[2 * 4 + 3];
    float pt_cam_x = cam2base[0 * 4 + 0] * tmp_pt[0] + cam2base[1 * 4 + 0] * tmp_pt[1] + cam2base[2 * 4 + 0] * tmp_pt[2];
    float pt_cam_y = cam2base[0 * 4 + 1] * tmp_pt[0] + cam2base[1 * 4 + 1] * tmp_pt[1] + cam2base[2 * 4 + 1] * tmp_pt[2];
    float pt_cam_z = cam2base[0 * 4 + 2] * tmp_pt[0] + cam2base[1 * 4 + 2] * tmp_pt[1] + cam2base[2 * 4 + 2] * tmp_pt[2];

    int volume_idx = pt_grid_z * voxel_grid_dim_y * voxel_grid_dim_x + pt_grid_y * voxel_grid_dim_x + pt_grid_x;
    if (pt_cam_z <= 0) {
      voxel_grid_TSDF[volume_idx] = -2.0f;
      continue;
    }

    int pt_pix_x = roundf(cam_K[0 * 3 + 0] * (pt_cam_x / pt_cam_z) + cam_K[0 * 3 + 2]);
    int pt_pix_y = roundf(cam_K[1 * 3 + 1] * (pt_cam_y / pt_cam_z) + cam_K[1 * 3 + 2]);
    if (pt_pix_x < 0 || pt_pix_x >= im_width || pt_pix_y < 0 || pt_pix_y >= im_height) {
      voxel_grid_TSDF[volume_idx] = -2.0f;
      continue;
    }

    float depth_val = depth_im[pt_pix_y * im_width + pt_pix_x];

    if (depth_val > 8) {
      voxel_grid_TSDF[volume_idx] = -2.0f;
      continue;
    }

    float diff = depth_val - pt_cam_z;

    // This is for labeling the -1 space (occluded space)
    if (diff < -0.1 || depth_val == 0.0) {
      voxel_grid_TSDF[volume_idx] = 2.0f;
      continue;
    }

    // This is for labeling the empty space
    if (diff > 0.1) {
      voxel_grid_TSDF[volume_idx] = -1.0f;
      continue;
    }

    // Integrate
    // float dist = fmin(1.0f, diff / trunc_margin);
    // float weight_old = voxel_grid_weight[volume_idx];
    // float weight_new = weight_old + 1.0f;
    // voxel_grid_weight[volume_idx] = weight_new;
    // voxel_grid_TSDF[volume_idx] = (voxel_grid_TSDF[volume_idx] * weight_old + dist) / weight_new;
    if (abs(diff) < 0.1) {
      voxel_grid_TSDF[volume_idx] = 1.0f;
    }
  }
}

// Loads a binary file with depth data and generates a TSDF voxel volume (5m x 5m x 5m at 1cm resolution)
// Volume is aligned with respect to the camera coordinates of the first frame (a.k.a. base frame)
int main(int argc, char * argv[]) {

  // Location of camera intrinsic file
  std::string cam_K_file = "data/camera-intrinsics.txt";
  std::string cam_origin_file = "data/origin/00017227_01e40e56e7c4006efc920560ac4d26b9_fl001_rm0004_0000.txt";
  std::string base2world_file = "data/camera/00017227_01e40e56e7c4006efc920560ac4d26b9_fl001_rm0004_0000.txt";
  std::string depth_im_file = "data/depth_real_png/00017227_01e40e56e7c4006efc920560ac4d26b9_fl001_rm0004_0000.png";
  std::string tsdf_bin_file = "tsdf.bin";

  // Location of folder containing RGB-D frames and camera pose files
  // std::string data_path = "data/rgbd-frames-yida";

  float cam_K[3 * 3];
  float cam_origin[3 * 1];
  float base2world[4 * 4];
  float cam2base[4 * 4];
  float cam2world[4 * 4];
  int im_width = 640;
  int im_height = 480;
  float depth_im[im_height * im_width];

  // Voxel grid parameters (change these to change voxel grid resolution, etc.)
  float voxel_grid_origin_x = 43.15f; // Location of voxel grid origin in base frame camera coordinates
  float voxel_grid_origin_y = 50.88f;
  float voxel_grid_origin_z = 0.05f;
  float voxel_size = 0.06f;
  float trunc_margin = 0.72f;//voxel_size * 5;
  int voxel_grid_dim_x = 80;
  int voxel_grid_dim_y = 80;
  int voxel_grid_dim_z = 48;

  // Manual parameters
  if (argc > 1) {
    cam_K_file = argv[1];
    cam_origin_file = argv[2];
    base2world_file = argv[3];
    depth_im_file = argv[4];
    tsdf_bin_file = argv[5];
  }

  // Read camera intrinsics
  std::vector<float> cam_K_vec = LoadMatrixFromFile(cam_K_file, 3, 3);
  std::copy(cam_K_vec.begin(), cam_K_vec.end(), cam_K);
  std::vector<float> cam_origin_vec = LoadMatrixFromFile(cam_origin_file, 3, 1);
  std::copy(cam_origin_vec.begin(), cam_origin_vec.end(), cam_origin);
  voxel_grid_origin_x = cam_origin[0];
  voxel_grid_origin_y = cam_origin[1];
  voxel_grid_origin_z = cam_origin[2];

  // Read base frame camera pose
  std::ostringstream base_frame_prefix;
  // base_frame_prefix << std::setw(6) << std::setfill('0') << base_frame_idx;
  // std::string base2world_file = data_path + "/frame-" + base_frame_prefix.str() + ".pose.txt";
  std::vector<float> base2world_vec = LoadMatrixFromFile(base2world_file, 4, 4);
  std::copy(base2world_vec.begin(), base2world_vec.end(), base2world);

  // Invert base frame camera pose to get world-to-base frame transform
  float base2world_inv[16] = {0};
  invert_matrix(base2world, base2world_inv);

  // Initialize voxel grid
  float * voxel_grid_TSDF = new float[voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z];
  for (int i = 0; i < voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z; ++i)
    voxel_grid_TSDF[i] = 0.0f;

  // Load variables to GPU memory
  float * gpu_voxel_grid_TSDF;
  hipMalloc(&gpu_voxel_grid_TSDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float));
  checkCUDA(__LINE__, hipGetLastError());
  hipMemcpy(gpu_voxel_grid_TSDF, voxel_grid_TSDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float), hipMemcpyHostToDevice);
  checkCUDA(__LINE__, hipGetLastError());
  float * gpu_cam_K;
  float * gpu_cam2base;
  float * gpu_depth_im;
  hipMalloc(&gpu_cam_K, 3 * 3 * sizeof(float));
  hipMemcpy(gpu_cam_K, cam_K, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&gpu_cam2base, 4 * 4 * sizeof(float));
  hipMalloc(&gpu_depth_im, im_height * im_width * sizeof(float));
  checkCUDA(__LINE__, hipGetLastError());

  // Loop through each depth frame and integrate TSDF voxel grid

    // std::ostringstream curr_frame_prefix;
    // curr_frame_prefix << std::setw(6) << std::setfill('0') << frame_idx;

    // // Read current frame depth
    // std::string depth_im_file = data_path + "/frame-" + curr_frame_prefix.str() + ".depth.png";
    ReadDepth(depth_im_file, im_height, im_width, depth_im);

    // Read base frame camera pose
    std::string cam2world_file = base2world_file; //data_path + "/frame-" + curr_frame_prefix.str() + ".pose.txt";
    std::vector<float> cam2world_vec = LoadMatrixFromFile(cam2world_file, 4, 4);
    std::copy(cam2world_vec.begin(), cam2world_vec.end(), cam2world);

    // Compute relative camera pose (camera-to-base frame)
    multiply_matrix(base2world_inv, cam2world, cam2base);

    // yida: here we should use base2world for rotation for alignment of the ground
    hipMemcpy(gpu_cam2base, base2world, 4 * 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_depth_im, depth_im, im_height * im_width * sizeof(float), hipMemcpyHostToDevice);
    checkCUDA(__LINE__, hipGetLastError());

    // std::cout << "Fusing: " << depth_im_file << std::endl;

    Integrate <<< voxel_grid_dim_z, voxel_grid_dim_y >>>(gpu_cam_K, gpu_cam2base, gpu_depth_im,
                                                         im_height, im_width, voxel_grid_dim_x, voxel_grid_dim_y, voxel_grid_dim_z,
                                                         voxel_grid_origin_x, voxel_grid_origin_y, voxel_grid_origin_z, voxel_size, trunc_margin,
                                                         gpu_voxel_grid_TSDF);

  // Load TSDF voxel grid from GPU to CPU memory
  hipMemcpy(voxel_grid_TSDF, gpu_voxel_grid_TSDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float), hipMemcpyDeviceToHost);
  // hipMemcpy(voxel_grid_weight, gpu_voxel_grid_weight, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float), hipMemcpyDeviceToHost);
  checkCUDA(__LINE__, hipGetLastError());

  // Compute surface points from TSDF voxel grid and save to point cloud .ply file
  // std::cout << "Saving surface point cloud (tsdf.ply)..." << std::endl;
  
  SaveVoxelGrid2SurfacePointCloud("tsdf.ply", voxel_grid_dim_x, voxel_grid_dim_y, voxel_grid_dim_z,
                                  voxel_size, voxel_grid_origin_x, voxel_grid_origin_y, voxel_grid_origin_z,
                                  voxel_grid_TSDF);

  // Save TSDF voxel grid and its parameters to disk as binary file (float array)
  // std::cout << "Saving TSDF voxel grid values to disk (tsdf.bin)..." << std::endl;
  std::ofstream outFile(tsdf_bin_file, std::ios::binary | std::ios::out);
  /*
  float voxel_grid_dim_xf = (float) voxel_grid_dim_x;
  float voxel_grid_dim_yf = (float) voxel_grid_dim_y;
  float voxel_grid_dim_zf = (float) voxel_grid_dim_z;
  outFile.write((char*)&voxel_grid_dim_xf, sizeof(float));
  outFile.write((char*)&voxel_grid_dim_yf, sizeof(float));
  outFile.write((char*)&voxel_grid_dim_zf, sizeof(float));
  outFile.write((char*)&voxel_grid_origin_x, sizeof(float));
  outFile.write((char*)&voxel_grid_origin_y, sizeof(float));
  outFile.write((char*)&voxel_grid_origin_z, sizeof(float));
  outFile.write((char*)&voxel_size, sizeof(float));
  outFile.write((char*)&trunc_margin, sizeof(float));
  */
  for (int i = 0; i < voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z; ++i) {
    outFile.write((char*)&voxel_grid_TSDF[i], sizeof(float));
  }
  outFile.close();

  return 0;
}
